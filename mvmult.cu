#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>

using namespace std;

template<typename T> T* flatten(T** M, int mWidth,int mHeight){

    T* result = (T*)malloc((mWidth*mHeight)*sizeof(T));

    for(int i = 0; i < mHeight; i++){

        memcpy(result + (i*mWidth),M[i],(mWidth*sizeof(T)));

    }

    return result;


}

__global__ void mvmult(float* M, float* v, int mvWidth, int mHeight, float* t){


    int tIndex = (blockIdx.x*blockDim.x) + threadIdx.x;

    int result = 0;
    
    #pragma unroll
    for(int i = 0; i < mHeight; i++){

        int index = (i*mHeight) + tIndex;

        result += M[index] * v[tIndex];

    }

    t[tIndex] = result;

    
    

    


    

}

int main(int argc, char** argv){

    int mvWidth = 3, mHeight = 3;

    //Host memory
    float **M, *Mf,*v, *t;

    //Device Memory
    float *M_d, *v_d, *t_d;

    v = (float*)malloc(mvWidth*sizeof(float));
    t = (float*)malloc(mvWidth*sizeof(float));

    M = (float**)malloc(mHeight*sizeof(float*));

    hipMalloc(&M_d,mvWidth*mHeight*sizeof(float));
    hipMalloc(&v_d,mvWidth*sizeof(float));
    hipMalloc(&t_d,mvWidth*sizeof(float));

    for(int i = 0; i < mHeight; i++){

        M[i] = (float*)malloc(mvWidth*sizeof(float));

        for(int j = 0; j < mvWidth; j++){

            M[i][j] = 3;

        }

    }

    for(int i = 0; i < mvWidth; i++){

        v[i] = 4;

    }

    Mf = flatten(M,mvWidth,mHeight);


    hipMemcpy(M_d,Mf,mvWidth*mHeight*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(v_d,v,mvWidth*sizeof(float),hipMemcpyHostToDevice);

    mvmult<<<1,mvWidth>>>(M_d,v_d,mvWidth,mHeight,t_d);

    hipMemcpy(t,t_d,mvWidth*sizeof(float),hipMemcpyDeviceToHost);

    free(Mf);
    free(v);
    
    hipFree(M_d);
    hipFree(v_d);
    hipFree(t_d);

    cout << "| " << t[0] << ' ' << t[1] << ' ' << t[2] << " |" << endl;

    free(t);

    for(int i = 0; i < mHeight; i++){

        free(M[i]);

    }

    free(M);
    free(t);

    return EXIT_SUCCESS;

}