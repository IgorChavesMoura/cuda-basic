#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void saxpy(int n, float a, float* x, float* y){

    int index = (blockIdx.x*blockDim.x) + threadIdx.x; //To understand this , remember how pointer works,
    // but now the pointer we need refers to the thread that will process the current array index. So we start in the first thread block and the step is the block dimension,
    // to get the block that contains the thread we need, and then we start at the beginning of that block and the index of the thread.
    //.x because the blocks has only 1 dimension
    //These variables types are dim3(a simple struct defined by CUDA with x, y, and z members).
    
    if(index < n){

        y[index] = a*x[index] + y[index];

    }

}

int main(){

    int N = 1<<20; //Converts 1 to binary and adds 20 zeros to it and convert to int again which results in 1048576

    float *x, *y; //Host memory space;
    
    float *d_x, *d_y; //Device memory space;

    //Standard host memory allocation
    x = (float*)malloc(N*sizeof(float));
    y = (float*)malloc(N*sizeof(float));

    //Device memory allocation, now the fun begins
    hipMalloc(&d_x, N*sizeof(float)); 
    hipMalloc(&d_y, N*sizeof(float));

    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    //Transfer content from host memory to device memory
    hipMemcpy(d_x,x,N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_y,y,N*sizeof(float),hipMemcpyHostToDevice);


    saxpy<<<(N + 255)/256,256>>>(N,2.0f,d_x,d_y);

    hipMemcpy(y,d_y,N*sizeof(float),hipMemcpyDeviceToHost);

    float maxError = 0.0f;

    for(int i = 0;i < N;i++){

        maxError = max(maxError, abs(y[i]-4.0f));

    }

    printf("Max error: %f\n", maxError);

    //Free memory on device
    hipFree(d_x);
    hipFree(d_y);

    //Free memory on host
    free(x);
    free(y);


}